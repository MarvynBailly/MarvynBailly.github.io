#include <iostream>
#include <hip/hip_runtime.h>

void printCudaDevices() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id) << "\n";
        std::cerr << "-> " << hipGetErrorString(error_id) << "\n";
        std::cout << "Result = FAIL\n";
        exit(EXIT_FAILURE);
    }

    std::cout << "Detected " << deviceCount << " CUDA capable device(s)\n";

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << "\n";
        std::cout << "  Total amount of global memory:                 " << (deviceProp.totalGlobalMem / 1048576.0f) << " MBytes\n";
        std::cout << "  (Multiprocessors: " << deviceProp.multiProcessorCount << ")\n";
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
        std::cout << "  Clock rate: " << deviceProp.clockRate * 1e-3f << " MHz\n";
        std::cout << "  Total amount of constant memory: " << deviceProp.totalConstMem << " bytes\n";
        std::cout << "  Total amount of shared memory per block: " << deviceProp.sharedMemPerBlock << " bytes\n";
        std::cout << "  Total number of registers available per block: " << deviceProp.regsPerBlock << "\n";
        std::cout << "  Warp size: " << deviceProp.warpSize << "\n";
        std::cout << "  Maximum number of threads per block: " << deviceProp.maxThreadsPerBlock << "\n";
        std::cout << "  Maximum sizes of each dimension of a block: " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << "\n";
        std::cout << "  Maximum sizes of each dimension of a grid: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << "\n";
    }
}

int main() {
    printCudaDevices();
    return 0;
}
