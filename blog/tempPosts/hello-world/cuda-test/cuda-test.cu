#include <iostream>
#include <hip/hip_runtime.h>

__global__ void testKernel(int* d_out) {
    int idx = threadIdx.x;
    d_out[idx] = idx;
}

int main() {
    const int arraySize = 256;
    int h_out[arraySize];

    int* d_out;
    hipMalloc((void**)&d_out, arraySize * sizeof(int));

    testKernel<<<1, arraySize>>>(d_out);
    hipMemcpy(h_out, d_out, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; ++i) {
        std::cout << "h_out[" << i << "] = " << h_out[i] << std::endl;
    }

    hipFree(d_out);
    return 0;
}
