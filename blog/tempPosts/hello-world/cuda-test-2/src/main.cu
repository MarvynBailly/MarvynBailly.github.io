#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function declaration
__global__ void addKernel(int* c, const int* a, const int* b);

void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " : " << hipGetErrorString(result) << std::endl;
        exit(result);
    }
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    checkCudaError(hipMalloc((void**)&dev_a, arraySize * sizeof(int)), "hipMalloc dev_a failed");
    checkCudaError(hipMalloc((void**)&dev_b, arraySize * sizeof(int)), "hipMalloc dev_b failed");
    checkCudaError(hipMalloc((void**)&dev_c, arraySize * sizeof(int)), "hipMalloc dev_c failed");

    checkCudaError(hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy a to dev_a failed");
    checkCudaError(hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice), "hipMemcpy b to dev_b failed");

    addKernel<<<1, arraySize>>>(dev_c, dev_a, dev_b);

    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    checkCudaError(hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy dev_c to c failed");

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
