#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Pheromone.h"

__global__ void diffuseKernel(Pheromone* grid, int rows, int cols, float diffusionRate, float evapSpeed, float dt) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < rows && y < cols) {
        float antSum = 0.0f;
        float foodSum = 0.0f;
        float antOriginalVal = grid[x * cols + y].oldAntConcentration;
        float foodOriginalVal = grid[x * cols + y].oldFoodConcentration;
        int count = 0;

        for (int dx = -1; dx <= 1; ++dx) {
            for (int dy = -1; dy <= 1; ++dy) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < rows && ny >= 0 && ny < cols) {
                    antSum += grid[nx * cols + ny].oldAntConcentration;
                    foodSum += grid[nx * cols + ny].oldFoodConcentration;
                    count++;
                }
            }
        }

        float antBlur = antSum / count;
        float foodBlur = foodSum / count;
        float antDiffused = antOriginalVal + diffusionRate * dt * (antBlur - antOriginalVal);
        float foodDiffused = foodOriginalVal + diffusionRate * dt * (foodBlur - foodOriginalVal);
        float antDiffusedAndEvap = max(0.0f, antDiffused - evapSpeed * dt);
        float foodDiffusedAndEvap = max(0.0f, foodDiffused - evapSpeed * dt);
        grid[x * cols + y].newAntConcentration = antDiffusedAndEvap;
        grid[x * cols + y].newFoodConcentration = foodDiffusedAndEvap;

        // keep feed smelly
        if(grid[x * cols + y].hasFood){
            grid[x * cols + y].newFoodConcentration = 1.0f;
        }

        // keep the base smelly
        if(grid[x * cols + y].hasColony){
            grid[x * cols + y].newAntConcentration = 1.0f;
        }
    }
}

__global__ void update(Pheromone* grid, int rows, int cols){
    // Swap new and old concentrations
    for (int i = 0; i < rows * cols; ++i) {
        grid[i].oldAntConcentration = grid[i].newAntConcentration;
        grid[i].oldFoodConcentration = grid[i].newFoodConcentration;
    }
}

extern "C" void launchDiffuseKernel(Pheromone* grid, int rows, int cols, float diffusionRate, float evapSpeed, float dt) {
    dim3 blockSize(16, 16);
    dim3 gridSize((rows + blockSize.x - 1) / blockSize.x, (cols + blockSize.y - 1) / blockSize.y);
    diffuseKernel<<<gridSize, blockSize>>>(grid, rows, cols, diffusionRate, evapSpeed, dt);
    update<<<gridSize, blockSize>>>(grid, rows, cols);
    hipDeviceSynchronize();
}
