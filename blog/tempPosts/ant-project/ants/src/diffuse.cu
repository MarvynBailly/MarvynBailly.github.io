#include "hip/hip_runtime.h"
#include "Pheromone.h"

// Kernel function declaration
__global__ void diffuseKernel(Pheromone* grid, Pheromone* newGrid, int rows, int cols, float diffusionRate, float evapSpeed, float dt);

__global__ void diffuseKernel(Pheromone* grid, Pheromone* newGrid, int rows, int cols, float diffusionRate, float evapSpeed, float dt) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < rows && y < cols) {
        float antSum = 0.0f;
        float foodSum = 0.0f;
        int count = 0;

        for (int dx = -1; dx <= 1; ++dx) {
            for (int dy = -1; dy <= 1; ++dy) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < rows && ny >= 0 && ny < cols) {
                    antSum += grid[nx * cols + ny].antConcentration;
                    foodSum += grid[nx * cols + ny].foodConcentration;
                    count++;
                }
            }
        }

        float antBlur = antSum / count;
        float foodBlur = foodSum / count;
        float antDiffused = antBlur * diffusionRate * dt;
        float foodDiffused = foodBlur * diffusionRate * dt;
        float antDiffusedAndEvap = max(0.0f, antDiffused - evapSpeed * dt);
        float foodDiffusedAndEvap = max(0.0f, foodDiffused - evapSpeed * dt);

        newGrid[x * cols + y].antConcentration = antDiffusedAndEvap;
        newGrid[x * cols + y].foodConcentration = foodDiffusedAndEvap;

        // Ensure colony and food cells maintain their pheromone concentration
        if (grid[x * cols + y].hasColony == true) {
            newGrid[x * cols + y].antConcentration = 1.0f;
        }
        if (grid[x * cols + y].hasFood) {
            newGrid[x * cols + y].foodConcentration = 1.0f;
        }
    }
}
