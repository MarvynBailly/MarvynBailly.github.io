#include "hip/hip_runtime.h"
#include "collision.h"

__global__ void detectCollisionsKernel(Particle* d_particles, int numParticles, int gridWidth, int gridHeight, int cellWidth, int cellHeight, float response_coef) {
    int cell_x = blockIdx.x;
    int cell_y = blockIdx.y;

    int start_x = cell_x * cellWidth;
    int start_y = cell_y * cellHeight;
    int end_x = start_x + cellWidth;
    int end_y = start_y + cellHeight;

    for (int i = 0; i < numParticles; ++i) {
        Particle& particle_1 = d_particles[i];
        if (particle_1.position_current.x >= start_x && particle_1.position_current.x < end_x &&
            particle_1.position_current.y >= start_y && particle_1.position_current.y < end_y) {
            
            // Check collisions within the cell and its neighbors
            for (int dx = -1; dx <= 1; ++dx) {
                for (int dy = -1; dy <= 1; ++dy) {
                    int neighbor_x = cell_x + dx;
                    int neighbor_y = cell_y + dy;
                    if (neighbor_x >= 0 && neighbor_x < gridWidth && neighbor_y >= 0 && neighbor_y < gridHeight) {
                        int neighbor_start_x = neighbor_x * cellWidth;
                        int neighbor_start_y = neighbor_y * cellHeight;
                        int neighbor_end_x = neighbor_start_x + cellWidth;
                        int neighbor_end_y = neighbor_start_y + cellHeight;

                        for (int j = 0; j < numParticles; ++j) {
                            Particle& particle_2 = d_particles[j];
                            if (particle_2.position_current.x >= neighbor_start_x && particle_2.position_current.x < neighbor_end_x &&
                                particle_2.position_current.y >= neighbor_start_y && particle_2.position_current.y < neighbor_end_y) {

                                if (&particle_1 == &particle_2) continue;

                                float dx = particle_1.position_current.x - particle_2.position_current.x;
                                float dy = particle_1.position_current.y - particle_2.position_current.y;
                                float dist2 = dx * dx + dy * dy;
                                float min_dist = particle_1.radius + particle_2.radius;
                                
                                if (dist2 < min_dist * min_dist) {
                                    float dist = sqrt(dist2);
                                    float nx = dx / dist;
                                    float ny = dy / dist;
                                    float mass_ratio_1 = particle_1.radius / (particle_1.radius + particle_2.radius);
                                    float mass_ratio_2 = particle_2.radius / (particle_1.radius + particle_2.radius);
                                    float delta = 0.5f * response_coef * (dist - min_dist);
                                    particle_1.position_current.x -= nx * (mass_ratio_2 * delta);
                                    particle_1.position_current.y -= ny * (mass_ratio_2 * delta);
                                    particle_2.position_current.x += nx * (mass_ratio_1 * delta);
                                    particle_2.position_current.y += ny * (mass_ratio_1 * delta);
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

void detectCollisions(Particle* d_particles, int numParticles, int gridWidth, int gridHeight, int cellWidth, int cellHeight, float response_coef, dim3 threadsPerBlock) {
    dim3 numBlocks(gridWidth, gridHeight);
    detectCollisionsKernel<<<numBlocks, threadsPerBlock>>>(d_particles, numParticles, gridWidth, gridHeight, cellWidth, cellHeight, response_coef);
    hipDeviceSynchronize();
}
